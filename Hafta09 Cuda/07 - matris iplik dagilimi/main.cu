#include "hip/hip_runtime.h"

#include<hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include "yardimci.hpp"
void matrisYazdir(int *C, const int nx, const int ny)
{
    int *ic = C;
    printf("\nMatrix: (%d.%d)\n", nx, ny);

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            printf("%3d", ic[ix]);

        }

        ic += nx;
        printf("\n");
    }

    printf("\n");
    return;
}

__global__ 
void iplikIndeksleriniYazdir(int *A, const int nx, const int ny)
{
    
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) global index"
           " %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y,
           ix, iy, idx, A[idx]);
    
}


int main(void)
{
    int nx = 8;
    int ny = 6;
    int elemanSayisi= nx*ny;

    int toplamBoyut = sizeof(int)*elemanSayisi;
    
    
    int *h_A = new int[toplamBoyut];
    for(int i=0;i<elemanSayisi;i++)
    {
        h_A[i]=i;
    }
    matrisYazdir(h_A,nx,ny);



    int *d_A;
    
    hipSetDevice(0);
    hipMalloc(&d_A,sizeof(int)*elemanSayisi);
    

    hipMemcpy(d_A,h_A,sizeof(int)*elemanSayisi,hipMemcpyHostToDevice);


    dim3 block(4,2);
    dim3 grid(((nx+block.x-1)/block.x),(ny+block.y-1)/block.y);
    printf("iplik Sayisi:%d\n",block.x);
    printf("block Sayisi:%d\n",grid.x);

    TIMERSTART(device);
    iplikIndeksleriniYazdir<<<grid,block>>>(d_A,nx,ny);
    hipDeviceSynchronize();
    TIMERSTOP(device)
    
    
    delete [] h_A;
 

    hipFree(d_A);
    return 0;
}