
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void kernel()
{
    const int threadId = threadIdx.x;
    printf("iplik %d 'dan Merhaba\n",threadId);
}
int main()
{
   
    hipSetDevice(0);

    kernel<<<1,4>>>();
    
    hipDeviceSynchronize();
    printf("Merhaba host\n");
}