#include "hip/hip_runtime.h"
#include "yardimci.hpp"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * simpleDivergence demonstrates divergent code on the GPU and its impact on
 * performance and CUDA metrics.
 */

__global__ void mathKernel1(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if (tid % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel2(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel3(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    bool ipred = (tid % 2 == 0);

    if (ipred)
    {
        ia = 100.0f;
    }

    if (!ipred)
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel4(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    int itid = tid >> 5;

    if (itid & 0x01 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void warmingup(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}


int main(int argc, char **argv)
{

    hipSetDevice(0);
    
    int size = 1<<12;
    int blocksize = 64;

    if(argc > 1) blocksize = atoi(argv[1]);

    if(argc > 2) size      = atoi(argv[2]);

    printf("Data size %d ", size);

   
    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

    // allocate gpu memory
    float *d_C;
    size_t nBytes = size * sizeof(float);
    hipMalloc((float**)&d_C, nBytes);


    hipDeviceSynchronize();
    TIMERSTART(warmup1)
    warmingup<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    TIMERSTOP(warmup1)



    TIMERSTART(mathKernel1)
    mathKernel1<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    TIMERSTOP(mathKernel1)
    
   
    TIMERSTART(mathKernel2)
    mathKernel2<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    TIMERSTOP(mathKernel2)
   
    TIMERSTART(mathKernel3)
    mathKernel3<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    TIMERSTOP(mathKernel3)
   

    TIMERSTART(mathKernel4)
    mathKernel4<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    TIMERSTOP(mathKernel4)

    // free gpu memory and reset divece
    hipFree(d_C);

    return EXIT_SUCCESS;
}