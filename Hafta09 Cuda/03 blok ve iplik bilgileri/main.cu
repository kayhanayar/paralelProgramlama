
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


__global__ 
void indexKontrol()
{
    printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) "
            "gridDim:(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z,
                blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
            gridDim.x,gridDim.y,gridDim.z);
}

int main(void)
{
    int eleman=6;

    dim3 block(3);


    dim3 grid= ((eleman+block.x-1)/block.x);

    // check grid and block dimension from host side
    printf("grid.x %d grid.y %d grid.z %d\n",grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n",block.x, block.y, block.z);
    indexKontrol<<<grid,block>>>();
    hipDeviceSynchronize();



    return 0;
}