#include "hip/hip_runtime.h"
#include "yardimci.hpp"
#include <hip/hip_runtime.h>
#include <stdio.h>



void ilkDegerleriAta(float *ip, const int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}

void cpuMatrisHesaplama(float *A, float *B, float *C, const int nx,
                     const int ny)
{
    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            C[ix] = A[ix] + B[ix];

        }

        A += nx;
        B += nx;
        C += nx;
    }
    return;
}


void sonuclariKontrolEt(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match)
        printf("Matrisler esit.\n\n");
    else
        printf("matrisler esit degil.\n\n");
}


__global__ void 
gpuMatrisToplama2D(float *MatA, float *MatB, float *MatC, 
                    int nx,
                    int ny)
{
    unsigned int sutun = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int satir = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int index = satir * nx + sutun;

    if (sutun < nx && satir < ny)
        MatC[index] = MatA[index] + MatB[index];
}

int main(int argc, char **argv)
{
    

    

    hipSetDevice(0);

    // set up data size of matrix
    int nx = 1 << 14;
    int ny = 1 << 14;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matris boyutu: nx %d ny %d\n", nx, ny);

    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;

    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);

    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    TIMERSTART(ilkDegerAtama)
    ilkDegerleriAta(h_A, nxy);
    ilkDegerleriAta(h_B, nxy);
    TIMERSTOP(ilkDegerAtama)

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);
    TIMERSTART(cpuileHesaplama)
    cpuMatrisHesaplama(h_A, h_B, hostRef, nx, ny);
    TIMERSTOP(cpuileHesaplama)

   
    float *d_MatA, *d_MatB, *d_MatC;
    hipMalloc((void **)&d_MatA, nBytes);
    hipMalloc((void **)&d_MatB, nBytes);
    hipMalloc((void **)&d_MatC, nBytes);

    
    hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);

    
    int dimx = 16;
    int dimy = 16;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    gpuMatrisToplama2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    TIMERSTART(GPU_ileHesaplama)
    gpuMatrisToplama2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    hipDeviceSynchronize();
    TIMERSTOP(GPU_ileHesaplama)
    printf("gpuMatrisToplama2D <<<(%d,%d), (%d,%d)>>> ", grid.x,
           grid.y,
           block.x, block.y);
   
    hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost);

    // check device results
    sonuclariKontrolEt(hostRef, gpuRef, nxy);

   
    hipFree(d_MatA);
    hipFree(d_MatB);
    hipFree(d_MatC);

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    return (0);
}