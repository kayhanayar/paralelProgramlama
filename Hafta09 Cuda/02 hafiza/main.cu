
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


__global__ 
void topla(int n, float *x, float *y)
{

  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1<<20; 
    
    
    float *h_x = new float[N];
    float *h_y = new float[N];
    float *h_sonuc = new float[N];
   
    for (int i = 0; i < N; i++) {
        h_x[i] = 1.0f;
        h_y[i] = 2.0f;
    }

    float *d_x,*d_y;
    hipMalloc(&d_x,sizeof(float)*N);
    hipMalloc(&d_y,sizeof(float)*N);

    hipMemcpy(d_x,h_x,sizeof(float)*N,hipMemcpyHostToDevice);
    hipMemcpy(d_y,h_y,sizeof(float)*N,hipMemcpyHostToDevice);

    topla<<<1,1>>>(N,d_x,d_y);
    hipDeviceSynchronize();

    hipMemcpy(h_sonuc,d_y,sizeof(float)*N,hipMemcpyDeviceToHost);

    for(int i=0;i<N;i++)
    {
        if(i%10==0)
            printf("\n");

        printf("%5.4f ",h_sonuc[i]);
    }


    delete [] h_x;
    delete [] h_y;

    hipFree(d_x);
    hipFree(d_y);
    return 0;
}