
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


__global__ 
void topla(int n, float *x, float *y)
{
    int i = threadIdx.x;
    y[i] = x[i] + y[i];
    printf("id:%d \n",i);
}

int main(void)
{
    int N = 32; 
    
    
    float *h_x = new float[N];
    float *h_y = new float[N];

   
    for (int i = 0; i < N; i++) {
        h_x[i] = 1.0f;
        h_y[i] = 2.0f;
    }

    float *d_x,*d_y;
    hipMalloc(&d_x,sizeof(float)*N);
    hipMalloc(&d_y,sizeof(float)*N);

    hipMemcpy(d_x,h_x,sizeof(float)*N,hipMemcpyHostToDevice);
    hipMemcpy(d_y,h_y,sizeof(float)*N,hipMemcpyHostToDevice);

    topla<<<1,32>>>(N,d_x,d_y);
    hipDeviceSynchronize();


    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(h_y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;    


    delete [] h_x;
    delete [] h_y;

    hipFree(d_x);
    hipFree(d_y);
    return 0;
}