#include "hip/hip_runtime.h"

#include<hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include "yardimci.hpp"


__global__ 
void toplaCihaz(int n, float *x, float *y,float* sonuc)
{
    
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i<n)
        sonuc[i] = pow(2,x[i]) + y[i]*x[i]+pow(2,y[i])+y[i]*x[i]+pow(2,y[i]);;
    
    
}

void toplaSunucu(int n,float *x, float *y,float *sonuc)
{
    for(int i=0;i<n;i++)
    {
        
        sonuc[i] = pow(2,x[i]) +y[i]*x[i]+pow(2,y[i])+y[i]*x[i]+pow(2,y[i]);
    }
}

void rastgeleDegerata(float *p,int size)
{
    for(int i=0;i<size;i++)
    {
        p[i] = (float)(rand()%0xFF)/10.0f;
    }
}

void sonuclariKontrolEt(float* host,float*device,int N)
{
    float epsilon = 1.0E-8;
    bool eslesme = 1;

    for(int i=0;i<N;i++)
    {
        if(abs(host[i]-device[i])>epsilon)
        {
            eslesme = 0;
            printf("Diziler esit degil\n");
            printf("%d indeksteki sunucu %5.4f gpu %5.4f",i,host[i],device[i]);
            break;
        }
    }
    if(eslesme)
        printf("Diziler esit");
}
int main(void)
{
    int N = 1UL<<24; 
    
    
    float *h_x = new float[N];
    float *h_y = new float[N];
    float *h_sonuc = new float[N];
    float *g_sonuc = new float[N];
   
    
    rastgeleDegerata(h_x,N);
    rastgeleDegerata(h_y,N);

    float *d_x,*d_y,*d_sonuc;
    
    hipSetDevice(0);
    hipMalloc(&d_x,sizeof(float)*N);
    hipMalloc(&d_y,sizeof(float)*N);
    hipMalloc(&d_sonuc,sizeof(float)*N);

    hipMemcpy(d_x,h_x,sizeof(float)*N,hipMemcpyHostToDevice);
    hipMemcpy(d_y,h_y,sizeof(float)*N,hipMemcpyHostToDevice);

    int iplikSayisi = 1024;
    dim3 block(iplikSayisi);
    dim3 grid((N+block.x-1)/block.x);
    printf("iplik Sayisi:%d\n",block.x);
    printf("block Sayisi:%d\n",grid.x);

    TIMERSTART(device);
    toplaCihaz<<<grid,block>>>(N,d_x,d_y,d_sonuc);
    hipDeviceSynchronize();
    TIMERSTOP(device)
    
    
    auto err=hipMemcpy(g_sonuc,d_sonuc,sizeof(float)*N,hipMemcpyDeviceToHost);
    if(err!=hipSuccess)
        printf("hata:%s",hipGetErrorString(err));
    
    TIMERSTART(host);
    toplaSunucu(N,h_x,h_y,h_sonuc);
    TIMERSTOP(host);
    sonuclariKontrolEt(h_sonuc,g_sonuc,N);
    


    delete [] h_x;
    delete [] h_y;
    delete [] h_sonuc;
    delete [] g_sonuc;

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_sonuc);
    return 0;
}